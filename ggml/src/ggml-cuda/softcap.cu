#include "hip/hip_runtime.h"
#include "softcap.cuh"

static __global__ void softcap_f32(const float * x, float * dst, float s_before, float s_after, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    float xi = s_before*x[i];
    dst[i] = s_after * tanh(xi);
}

static void softcap_f32_cuda(const float * x, float * dst, float s_before, float s_after, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SOFTCAP_BLOCK_SIZE - 1) / CUDA_SOFTCAP_BLOCK_SIZE;
    softcap_f32<<<num_blocks, CUDA_SOFTCAP_BLOCK_SIZE, 0, stream>>>(x, dst, s_before, s_after, k);
}

void ggml_cuda_op_softcap(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float scales[2];
    memcpy(scales, dst->op_params, sizeof(scales));

    softcap_f32_cuda(src0_d, dst_d, scales[0], scales[1], ggml_nelements(src0), stream);
}
