#include "hip/hip_runtime.h"
#include "iqk_mmvq.cuh"

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs);

namespace {
template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__((ncols_y <= 4 ? 4 : 2)*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__global__ void iqk_mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int nwarps              = ncols_y <= 4 ? 4 : 2;
    constexpr int rows_per_cuda_block = ncols_y == 1 ? 1 : 2;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda(vx, &y[j*blocks_per_col_y + kby], (row0 + i)*blocks_per_row_x + kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            dst[j*nrows_dst + row0 + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda>
void iqk_mul_mat_vec_q_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    GGML_ASSERT(ncols_x % ggml_blck_size(type) == 0);
    //GGML_ASSERT(ncols_y <= MMVQ_MAX_BATCH_SIZE);

    int id = ggml_cuda_get_device();

    int64_t nwarps = 1;
    int64_t rows_per_cuda_block = 1;

    if (ggml_cuda_info().devices[id].cc < CC_RDNA2) { // NVIDIA and AMD older than RDNA2
        switch(ncols_y) {
            case 1:
                nwarps = 4;
                rows_per_cuda_block = 1;
                break;
            case 2:
            case 3:
            case 4:
                nwarps = 4;
                rows_per_cuda_block = 2;
                break;
            case 5:
            case 6:
            case 7:
            case 8:
                nwarps = 2;
                rows_per_cuda_block = 2;
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
    }
    const int64_t nblocks = (nrows_x + rows_per_cuda_block - 1) / rows_per_cuda_block;
    const dim3 block_nums(nblocks, 1, 1);
    const dim3 block_dims(WARP_SIZE, nwarps, 1);

    switch (ncols_y) {
        case 1:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 1><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 2:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 2><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 3:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 3><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 4:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 4><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 5:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 5><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 6:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 6><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 7:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 7><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        case 8:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 8><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

__device__ __forceinline__ void get_int_from_table_16_shift(const uint32_t & q4, uint16_t shift, const uint8_t * all_values,
        int & val1, int & val2) {

    uint32_t aux32; const uint8_t * q8 = (const uint8_t *)&aux32;
    aux32 = q4 & 0x0f0f0f0f;
    const uint8_t * values = all_values + 16*(shift & 1);
    uint16_t v1 = values[q8[0]] | (values[q8[1]] << 8);
    uint16_t v2 = values[q8[2]] | (values[q8[3]] << 8);
    val1 = v1 | (v2 << 16);
    aux32 = (q4 >> 4) & 0x0f0f0f0f;
    values = all_values + 8*(shift & 2);
    v1 = values[q8[0]] | (values[q8[1]] << 8);
    v2 = values[q8[2]] | (values[q8[3]] << 8);
    val2 = v1 | (v2 << 16);
}

#define VDR_IQ4_K_Q8_1_MMVQ 4
#define VDR_IQ4_K_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq4_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    const block_iq4_k * bq4 = (const block_iq4_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq4k_values;

    // iqs is 0...28
    const int ib32 = iqs/4;
    // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint16_t * q4 = (const uint16_t *)bq4->qs + 8*ib32;
    const uint16_t extra = bq4->extra >> 2*ib32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        const uint32_t aux32 = q4[2*j+0] | (q4[2*j+1] << 16);
        get_int_from_table_16_shift(aux32, extra, all_values, v1, v2);
        sumi1 = ggml_cuda_dp4a(v1, q8[j+0], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8[j+4], sumi2);
    }
    const float d = __half2float(bq4->d) * __low2float(bq8_1[ib32].ds);
    const uint8_t sh = bq4->scales_h[ib32/2] >> 4*(ib32%2);
    const int ls1 = ((bq4->scales_l[ib32] & 0xf) | ((sh << 4) & 0x30)) - 32;
    const int ls2 = ((bq4->scales_l[ib32] >>  4) | ((sh << 2) & 0x30)) - 32;
    return d * (sumi1 * ls1 + sumi2 * ls2);
}

#define VDR_IQ5_K_Q8_1_MMVQ 4
#define VDR_IQ5_K_Q8_1_MMQ  4

__device__ __forceinline__ int int_from_table(const uint8_t * a8, const uint8_t * values) {
    uint16_t v1 = values[a8[0]] | (values[a8[1]] << 8);
    uint16_t v2 = values[a8[2]] | (values[a8[3]] << 8);
    return v1 | (v2 << 16);
}

__device__ __forceinline__ float vec_dot_iq5_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {


    const block_iq5_k * bq5 = (const block_iq5_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq5nl_values;

    int i4 = iqs/4;  // 0...7.  Blocks of 16 index is 4*(i4/2) + (i4%2) + (0 and 2)

    const int32_t  * q8_1 = (const int *)bq8_1[2*(i4/2)+0].qs + 4*(i4%2);
    const int32_t  * q8_2 = (const int *)bq8_1[2*(i4/2)+1].qs + 4*(i4%2);
    const uint32_t * q4 = (const uint32_t *)bq5->qs + 8*(i4/2) + 4*(i4%2);
    const uint32_t * qh = (const uint32_t *)bq5->qh + 4*(i4%2);
    const uint16_t extra = bq5->extra >> (4*(i4/2) + (i4%2));
    const uint8_t * values1 = all_values + 32*(extra & 1);
    const uint8_t * values2 = all_values +  8*(extra & 4);
    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t h = qh[j] >> 2*(i4/2);
        aux32[0] = ((q4[j] >> 0) & 0x0f0f0f0f) | ((h << 4) & 0x10101010);
        aux32[1] = ((q4[j] >> 4) & 0x0f0f0f0f) | ((h << 3) & 0x10101010);
        v1 = int_from_table(a8+0, values1);
        v2 = int_from_table(a8+4, values2);
        sumi1 = ggml_cuda_dp4a(v1, q8_1[j], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8_2[j], sumi2);
    }
    const float d5 = __half2float(bq5->d);
    const uint8_t sh = bq5->scales_h[i4/2] >> 2*(i4%2);
    const int ls1 = (((bq5->scales_l[2*(i4/2)+0] >> 4*(i4%2)) & 0xf) | ((sh << 4) & 0x30)) - 32;
    const int ls2 = (((bq5->scales_l[2*(i4/2)+1] >> 4*(i4%2)) & 0xf) | ((sh << 0) & 0x30)) - 32;
    return d5 * (__low2float(bq8_1[2*(i4/2)+0].ds) * sumi1 * ls1 + __low2float(bq8_1[2*(i4/2)+1].ds) * sumi2 * ls2);
}

#define VDR_IQ2_K_Q8_1_MMVQ 4
#define VDR_IQ2_K_Q8_1_MMQ  4

// TODO
__device__ __forceinline__ float vec_dot_iq2_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {
    return 0;
//
//    const block_iq2_k * bq4 = (const block_iq2_k *) vbq + kbx;
//    const uint8_t * all_values = (const uint8_t *)iq4k_values;
//
//    // iqs is 0...28
//    const int ib32 = iqs/4;
//    // Why iqs/4 ?
//    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
//    const uint16_t * q4 = (const uint16_t *)bq4->qs + 8*ib32;
//    const uint16_t extra = bq4->extra >> 2*ib32;
//    int v1, v2;
//    int sumi1 = 0, sumi2 = 0;
//    for (int j = 0; j < 4; ++j) {
//        const uint32_t aux32 = q4[2*j+0] | (q4[2*j+1] << 16);
//        get_int_from_table_16_shift(aux32, extra, all_values, v1, v2);
//        sumi1 = ggml_cuda_dp4a(v1, q8[j+0], sumi1);
//        sumi2 = ggml_cuda_dp4a(v2, q8[j+4], sumi2);
//    }
//    const float d = __half2float(bq4->d) * __low2float(bq8_1[ib32].ds);
//    const uint8_t sh = bq4->scales_h[ib32/2] >> 4*(ib32%2);
//    const int ls1 = ((bq4->scales_l[ib32] & 0xf) | ((sh << 4) & 0x30)) - 32;
//    const int ls2 = ((bq4->scales_l[ib32] >>  4) | ((sh << 2) & 0x30)) - 32;
//    return d * (sumi1 * ls1 + sumi2 * ls2);
}

}

void mul_mat_vec_iq2_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K, VDR_IQ2_K_Q8_1_MMVQ, vec_dot_iq2_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq4_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_K, VDR_IQ4_K_Q8_1_MMVQ, vec_dot_iq4_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

void mul_mat_vec_iq5_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_K, VDR_IQ5_K_Q8_1_MMVQ, vec_dot_iq5_k_q8_1>(vx, vy, dst, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, stream);
}

